#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 1<<30;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  hipDeviceEnablePeerAccess(1, 0);
  printf("Warmup \n");
  hipSetDevice(0);
  saxpy<<<(N+255)/256, 256>>>(N, 1.0f, d_x, d_y);
  hipDeviceSynchronize();
  hipSetDevice(1);
  hipDeviceEnablePeerAccess(0, 0);
  saxpy<<<(N+255)/256, 256>>>(N, 1.0f, d_x, d_y);
  hipDeviceSynchronize();
  printf("Warmup done\n");


  // experiment 1: kernel0 on gpu 0, kernel 1 on gpu 1, data dependency via peer access
  hipSetDevice(0);
  hipStream_t stream0;
  hipStreamCreate(&stream0);
  hipEvent_t done0; 
  hipEventCreateWithFlags(&done0, hipEventDisableTiming);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256, 0, stream0>>>(N, 2.0f, d_x, d_y);
  hipEventRecord(done0, stream0);

  hipDeviceEnablePeerAccess(0, 0);   // Enable peer-to-peer access

  // launch another kernel on gpu 1
  hipSetDevice(1);
  hipStream_t stream1;
  hipStreamCreate(&stream1);
  hipStreamWaitEvent(stream1, done0, 0);
  for (int i = 0; i < 2; i++){
    saxpy<<<(N+255)/256, 256, 0, stream1>>>(N, 1.0f, d_x, d_y);
  }

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-8.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}