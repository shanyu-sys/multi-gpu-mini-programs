#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y, *d_x_managed, *d_y_managed;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMallocManaged(&d_x_managed, N*sizeof(float)); 
  hipMallocManaged(&d_y_managed, N*sizeof(float));
  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 0.0f;
  }

  hipMemcpy(d_x_managed, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y_managed, y, N*sizeof(float), hipMemcpyHostToDevice);
  
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
  
  hipDeviceEnablePeerAccess(1, 0);
  printf("Warmup \n");
  hipSetDevice(0);
  saxpy<<<(N+255)/256, 256>>>(N, 1.0f, d_x, d_y);
  hipDeviceSynchronize();
  hipSetDevice(1);
  hipDeviceEnablePeerAccess(0, 0);
  saxpy<<<(N+255)/256, 256>>>(N, 1.0f, d_x, d_y);
  hipDeviceSynchronize();
  printf("Warmup done\n");

  printf("benchmark peer access\n");
  for (int i = 0; i < 3; i++) {
    hipSetDevice(0);
    // hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
    // Perform SAXPY on 1M elements
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float milliseconds = 0;
    for (int j = 0; j < 3; j++) {
      hipEventRecord(start);
      saxpy<<<(N+255)/256, 256>>>(N, 1.0f, d_x, d_y);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipDeviceSynchronize();
      hipEventElapsedTime(&milliseconds, start, stop);
      printf("Device 0 kernel time: %f ms\n", milliseconds);
    }

    hipSetDevice(1);
    hipDeviceEnablePeerAccess(0, 0);
    for (int j = 0; j < 3; j++) {
      hipEventRecord(start);
      saxpy<<<(N+255)/256, 256>>>(N, 1.0f, d_x, d_y);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipDeviceSynchronize();
      hipEventElapsedTime(&milliseconds, start, stop);
      printf("Device 1 kernel time: %f ms\n", milliseconds);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    float max_value = 0.0f;
    for (int i = 0; i < N; i++) {
      if (y[i] > max_value) max_value = y[i];
    }
    printf("Max value: %f\n", max_value);
    hipDeviceDisablePeerAccess(0);
  }

  printf("benchmark managed\n");
  for (int i = 0; i < 3; i++) {
    hipSetDevice(0);
    // hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
    // Perform SAXPY on 1M elements
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float milliseconds = 0;
    for (int j = 0; j < 3; j++) {
      hipEventRecord(start);
      saxpy<<<(N+255)/256, 256>>>(N, 1.0f, d_x_managed, d_y_managed);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds, start, stop);
      printf("Device 0 kernel time: %f ms\n", milliseconds);
    }

    hipSetDevice(1);
    // hipDeviceEnablePeerAccess(0, 0);
    for (int j = 0; j < 3; j++) {
      hipEventRecord(start);
      saxpy<<<(N+255)/256, 256>>>(N, 1.0f, d_x_managed, d_y_managed);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds, start, stop);
      printf("Device 1 kernel time: %f ms\n", milliseconds);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(y, d_y_managed, N*sizeof(float), hipMemcpyDeviceToHost);

    float max_value = 0.0f;
    for (int i = 0; i < N; i++) {
      if (y[i] > max_value) max_value = y[i];
    }
    printf("Max value: %f\n", max_value);
  }

  printf("benchmark managed with prefetch\n");
  for (int i = 0; i < 3; i++) {
    hipSetDevice(0);
    // hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
    // Perform SAXPY on 1M elements
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float milliseconds = 0;
    for (int j = 0; j < 3; j++) {
      hipEventRecord(start);
      saxpy<<<(N+255)/256, 256>>>(N, 1.0f, d_x_managed, d_y_managed);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds, start, stop);
      printf("Device 0 kernel time: %f ms\n", milliseconds);
    }

    hipSetDevice(1);
    // hipDeviceEnablePeerAccess(0, 0);
    hipMemPrefetchAsync(d_x_managed, N*sizeof(float), 1);
    hipMemPrefetchAsync(d_y_managed, N*sizeof(float), 1);
    for (int j = 0; j < 3; j++) {
      hipEventRecord(start);
      saxpy<<<(N+255)/256, 256>>>(N, 1.0f, d_x_managed, d_y_managed);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds, start, stop);
      printf("Device 1 kernel time: %f ms\n", milliseconds);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(y, d_y_managed, N*sizeof(float), hipMemcpyDeviceToHost);

    float max_value = 0.0f;
    for (int i = 0; i < N; i++) {
      if (y[i] > max_value) max_value = y[i];
    }
    printf("Max value: %f\n", max_value);
  }

  // sleep 5 seconds
  usleep(5000000);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}