#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 1<<30;
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));


  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  hipSetDevice(0);
  hipStream_t stream0;
  hipStreamCreate(&stream0);
  hipEvent_t done0; 
  hipEventCreateWithFlags(&done0, hipEventDisableTiming);

  for (int i = 0; i < 2; i++){
    saxpy<<<numBlocks, blockSize, 0, stream0>>>(N, 2.0f, x, y);
  }
  hipEventRecord(done0, stream0);

  hipDeviceEnablePeerAccess(0, 0);   // Enable peer-to-peer access

  // launch another kernel on gpu 1
  hipSetDevice(1);
  hipStream_t stream1;
  hipStreamCreate(&stream1);
  hipStreamWaitEvent(stream1, done0, 0);

  hipMemPrefetchAsync(x, N*sizeof(float), 1);
  for (int i = 0; i < 2; i++){
    saxpy<<<(N+255)/256, 256, 0, stream1>>>(N, 1.0f, x, y);
  }
  hipDeviceSynchronize();

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-8.0f));
  printf("Max error: %f\n", maxError);

  hipFree(x);
  hipFree(y); 
}