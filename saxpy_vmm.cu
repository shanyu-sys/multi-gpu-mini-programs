#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  // Initialize CUDA driver API
  hipInit(0);
  int N = 1<<30;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  size_t size = N * sizeof(float);
  size_t granularity;
  
  // Get memory allocation properties
  hipMemAllocationProp prop = {};
  prop.type = hipMemAllocationTypePinned;
  prop.location.type = hipMemLocationTypeDevice;
  prop.location.id = 0; // GPU 0
  
  hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);
  
  // Round up to granularity
  size_t padded_size = ((size + granularity - 1) / granularity) * granularity;
  
  // Allocate virtual address range
  hipDeviceptr_t d_x_ptr, d_y_ptr;
  hipMemAddressReserve(&d_x_ptr, padded_size, 0, 0, 0);
  hipMemAddressReserve(&d_y_ptr, padded_size, 0, 0, 0);
  
  // Create physical memory handle
  hipMemGenericAllocationHandle_t handle_x, handle_y;
  hipMemCreate(&handle_x, padded_size, &prop, 0);
  hipMemCreate(&handle_y, padded_size, &prop, 0);
  
  // Map virtual addresses to physical memory
  hipMemMap(d_x_ptr, padded_size, 0, handle_x, 0);
  hipMemMap(d_y_ptr, padded_size, 0, handle_y, 0);
  
  // Set access for both GPUs
  hipMemAccessDesc accessDesc[2];
  accessDesc[0].location.type = hipMemLocationTypeDevice;
  accessDesc[0].location.id = 0;
  accessDesc[0].flags = hipMemAccessFlagsProtReadWrite;
  
  accessDesc[1].location.type = hipMemLocationTypeDevice;
  accessDesc[1].location.id = 1;
  accessDesc[1].flags = hipMemAccessFlagsProtReadWrite;
  
  hipMemSetAccess(d_x_ptr, padded_size, accessDesc, 2);
  hipMemSetAccess(d_y_ptr, padded_size, accessDesc, 2);
  
  d_x = (float*)d_x_ptr;
  d_y = (float*)d_y_ptr;

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  printf("Warmup \n");
  hipSetDevice(0);
  saxpy<<<(N+255)/256, 256>>>(N, 1.0f, d_x, d_y);
  hipDeviceSynchronize();
  hipSetDevice(1);
  saxpy<<<(N+255)/256, 256>>>(N, 1.0f, d_x, d_y);
  hipDeviceSynchronize();
  printf("Warmup done\n");


  // experiment 1: kernel0 on gpu 0, kernel 1 on gpu 1, data dependency via VMM
  hipSetDevice(0);
  hipStream_t stream0;
  hipStreamCreate(&stream0);
  hipEvent_t done0; 
  hipEventCreateWithFlags(&done0, hipEventDisableTiming);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256, 0, stream0>>>(N, 2.0f, d_x, d_y);
  hipEventRecord(done0, stream0);

  // launch another kernel on gpu 1
  hipSetDevice(1);
  hipStream_t stream1;
  hipStreamCreate(&stream1);
  hipStreamWaitEvent(stream1, done0, 0);
  for (int i = 0; i < 2; i++){
    saxpy<<<(N+255)/256, 256, 0, stream1>>>(N, 1.0f, d_x, d_y);
  }
  hipDeviceSynchronize();

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-8.0f));
  printf("Max error: %f\n", maxError);

  // Cleanup VMM resources
  hipMemUnmap(d_x_ptr, padded_size);
  hipMemUnmap(d_y_ptr, padded_size);
  hipMemRelease(handle_x);
  hipMemRelease(handle_y);
  hipMemAddressFree(d_x_ptr, padded_size);
  hipMemAddressFree(d_y_ptr, padded_size);
  
  free(x);
  free(y);
}